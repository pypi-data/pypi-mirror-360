#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchDecodeParams<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

using ParamsMlaT = BatchDecodeParamsMLA<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<128, 16, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, ParamsMlaT>(
    ParamsMlaT params,
    __hip_fp8_e4m3_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

}
    