#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SingleDecodeParams<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz>;

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e4m3_fnuz* tmp,
    hipStream_t stream);

}
    