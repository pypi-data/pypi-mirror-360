#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SinglePrefillParams<half, __hip_fp8_e4m3_fnuz, half>;

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

template hipError_t SinglePrefillWithKVCacheDispatched<256, 256, PosEncodingMode::kNone, 0, MaskMode::kCausal, DefaultAttention<
    false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    half* tmp,
    hipStream_t stream);

}
    