#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = BatchDecodeParams<__hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

template hipError_t BatchDecodeWithPagedKVCacheDispatched<256, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

using ParamsMlaT = BatchDecodeParamsMLA<__hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz, int32_t>;

template hipError_t BatchDecodeWithPagedKVCacheDispatchedMLA<256, 32, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, ParamsMlaT>(
    ParamsMlaT params,
    __hip_fp8_e5m2_fnuz* tmp_v, float* tmp_s,
    hipStream_t stream);

}
    