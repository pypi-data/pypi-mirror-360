#include "hip/hip_runtime.h"
#include <flashinfer/attention_impl.cuh>

namespace flashinfer {

using Params = SingleDecodeParams<__hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz, __hip_fp8_e5m2_fnuz>;

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/false, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/false, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp,
    hipStream_t stream);

template hipError_t SingleDecodeWithKVCacheDispatched<128, PosEncodingMode::kNone, DefaultAttention<
    /*use_custom_mask=*/false, /*use_sliding_window=*/true, /*use_logits_soft_cap=*/true, /*use_alibi_bias=*/false>, Params>(
    Params params,
    __hip_fp8_e5m2_fnuz* tmp,
    hipStream_t stream);

}
    